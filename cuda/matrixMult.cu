#include "hip/hip_runtime.h"
// Cmeated By:      Jacob Huckins & Mikey Thoreson
// Last Modified:   03/04/2025

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <jpeglib.h>

const int WINDOW_DIM = 9;
const int PIC_WIDTH = 32;
const int PIC_HEIGHT = 32;

// implementation taken from pg 109 of nvidia cuda C programming guide ver 4.2
__device__ double doubleAtomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull=
		(unsigned long long int*) address;
	unsigned long long int old=*address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val +
					__longlong_as_double(assumed)));
	} while (assumed!= old);
	return __longlong_as_double(old);
}

__device__ void winDotProduct(int *l_pic, int *r_pic, int x_center, int y_center, int offset, double &retVal){
	int half_win = (WINDOW_DIM / 2);
	int l_idx = (x_center - half_win) + ((y_center - half_win) * PIC_WIDTH);
	int r_idx = (x_center + offset - half_win) + ((y_center  - half_win) * PIC_WIDTH);
	int l_idx_win = 0;
	int r_idx_win = 0; 
	retVal = 0;

	for(int x = 0; x < WINDOW_DIM; x++){
		for(int y = 0; y < WINDOW_DIM; y++){
			l_idx_win = l_idx + (x + (y * PIC_WIDTH));
			r_idx_win = r_idx + (x + (y * PIC_WIDTH));
			retVal += l_pic[l_idx_win] * r_pic[r_idx_win];
		}
	}
}

__device__ void windowSum(int *pic, int x_center, int y_center, double &retVal){
	int half_win = WINDOW_DIM / 2;
	int idx = (x_center - half_win) + ((y_center - half_win) * PIC_WIDTH);
	int idx_win = 0;

	retVal = 0;

	for(int x= 0; x < WINDOW_DIM; x++){
		for(int y = 0; y < WINDOW_DIM; y++){
			idx_win = idx + (x + (y * PIC_WIDTH));
			retVal += pic[idx_win];
		}
	}
}

__global__ void correlationCoefficient(int *l, int *r, int row, double *out)
{
	int x; x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y; y = threadIdx.y + (blockDim.y * blockIdx.y);

	if(!(x - (WINDOW_DIM / 2) < 0) && !(x + (WINDOW_DIM / 2) >= PIC_WIDTH)){
		if(!(y - (WINDOW_DIM / 2) < 0) && !(y + (WINDOW_DIM / 2) >= PIC_HEIGHT)){
			double N = WINDOW_DIM * WINDOW_DIM;

			// calc L dot 1
			double Ld1;
			windowSum(l, x, row, Ld1);

			// calc R dot 1
			double Rd1;
			windowSum(r, x + y, row, Rd1);

			// calc (L dot R) / N
			double LdR;
			winDotProduct(l, r, x, row, y, LdR);

			// calc (L dot L) / N
			double LdL;
			winDotProduct(l, l, x, row, 0, LdL);

			// calc (R dot R) / N
			double RdR;
			winDotProduct(r, r, x+y, row, 0, RdR);
			//winDotProduct(r, r, 40+y, 40, 0, LdR);

			// calculate correlation coefficient
			// [n(X.Y) - (X.1)(Y.1)] / [(n(X.X) - X.1)(n(Y.Y - Y.1))]
			//double top = ((N) * LdR) - (Ld1 * Rd1);
			double top = (LdR/N) - ((Ld1/N)*(Rd1/N));
			//double bot = ((N * LdL) - Ld1) * (N * (RdR - Rd1));
			double bot = sqrtf(
					(LdL/N) - ((Ld1/N)*(Ld1/N))
					) *
				sqrtf(
						(RdR/N) - ((Rd1/N)*(Rd1/N))
				     );

			__syncthreads();
			out[x + (y*PIC_WIDTH)] = (top/bot);
		}
	}
	else
	{
		out[x + (y*PIC_WIDTH)] = 0.0;
	}
}
//image utilities

PPMImage* readPPM(const char* filename, int type) {
	FILE* file = fopen(filename, "rb");
	int numBytes;

	if (!file) {
		perror("Error opening file");
		exit(0);
	}

	if(type==0) numBytes=1; //grayscale

int main()
{
	//    my reference code
	//    int* helloMtx = (int*) malloc(sizeof(int) * 9);
	//    int* d_helloMtx;
	//    size_t size = 9 * sizeof(int);
	//
	//    hipMalloc(&d_helloMtx, 9 * sizeof(int));
	//    helloWorld<<<1,9>>>(d_helloMtx);
	//    hipMemcpy(helloMtx, d_helloMtx, size, hipMemcpyDeviceToHost);
	//    hipFree(&d_helloMtx);
	//    for(int i = 0; i < 9; i++){
	//        printf("%d", helloMtx[i]);
	//    }

	int* leftimg = (int*) malloc(sizeof(int) * PIC_WIDTH * PIC_HEIGHT);
	int* rightimg = (int*) malloc(sizeof(int) * PIC_WIDTH * PIC_HEIGHT);
	double* h_CorrCoefMtx = (double*) malloc(sizeof(double) * PIC_WIDTH * PIC_HEIGHT); 
	for(int i = 0; i < PIC_WIDTH * PIC_HEIGHT; i++){
			leftimg[i] = i;
			rightimg[i] = (i*i)%255;
			h_CorrCoefMtx[i] = 0.0;
	}

	int picSize = PIC_WIDTH * PIC_HEIGHT;

	int * d_leftimg;
	hipMalloc(&d_leftimg, sizeof(int) * picSize);
	int * d_rightimg;
	hipMalloc(&d_rightimg, sizeof(int) * picSize);
	double * d_CC;
	hipMalloc(&d_CC, sizeof(double) * picSize);

	// setup the 2d matrices that will hold the result of our matrix
	// mult operations, for each combination of pixels on each pixel

	hipMemcpy(d_leftimg, leftimg, sizeof(int) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyHostToDevice);
	hipMemcpy(d_rightimg, rightimg, sizeof(int) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyHostToDevice);
	hipMemcpy(d_CC, h_CorrCoefMtx, sizeof(double) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyHostToDevice);

	dim3 threadCount(32,32);
	dim3 blockCount(1,1);
	correlationCoefficient<<<blockCount, threadCount>>>(d_leftimg, d_rightimg, 20, d_CC);

	hipMemcpy(h_CorrCoefMtx, d_CC, sizeof(double) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyDeviceToHost);


	for(int i = 0; i < PIC_WIDTH; i++){
		for(int j = 0; j < PIC_HEIGHT; j++){
			printf("%f ", h_CorrCoefMtx[i + (j*PIC_WIDTH)]);
		}
		printf("\n");
	}

	hipFree(&d_leftimg);
	hipFree(&d_rightimg);
	hipFree(&d_CC);
	return 0;
}
