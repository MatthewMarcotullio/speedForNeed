// Cmeated By:      Jacob Huckins & Mikey Thoreson
// Last Modified:   03/04/2025

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


const int WINDOW_DIM = 9;
const int PIC_WIDTH = 81;
const int PIC_HEIGHT = 81;

// implementation taken from pg 109 of nvidia cuda C programming guide ver 4.2
__device__ double doubleAtomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull=
		(unsigned long long int*) address;
	unsigned long long int old=*address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val +
					__longlong_as_double(assumed)));
	} while (assumed!= old);
	return __longlong_as_double(old);
}

__device__ void winDotProduct(int *l_pic, int *r_pic, int x_center, int y_center, int offset, double &retVal){
	int half_win = (WINDOW_DIM / 2);
	int l_idx = (x_center - half_win) + ((y_center - half_win) * PIC_WIDTH);
	int r_idx = (x_center + offset - half_win) + ((y_center  - half_win) * PIC_WIDTH);
	int l_idx_win = 0;
	int r_idx_win = 0;

	retVal = 0;

	for(int x = 0; x < WINDOW_DIM; x++){
		for(int y = 0; y < WINDOW_DIM; y++){
			l_idx_win = l_idx + (x + (y * PIC_WIDTH));
			r_idx_win = r_idx + (x + (y * PIC_WIDTH));
			retVal += l_pic[l_idx_win] * r_pic[r_idx_win];
		}
	}
}

__device__ void windowSum(int *pic, int x_center, int y_center, double &retVal){
	int half_win = WINDOW_DIM / 2;
	int idx = (x_center - half_win) + ((y_center - half_win) * PIC_WIDTH);
	int idx_win = 0;

	retVal = 0;

	for(int x= 0; x < WINDOW_DIM; x++){
		for(int y = 0; y < WINDOW_DIM; y++){
			idx_win = idx + (x + (y * PIC_WIDTH));
			retVal += pic[idx_win];
		}
	}
}

__global__ void correlationCoefficient(int *l, int *r, int row, double *out)
{
	int x = threadIdx.x + (blockDim.x * blockIdx.x);
	int y = threadIdx.y + (blockDim.y * blockIdx.y);

	//	if(!(x - (WINDOW_DIM / 2) < 0) && !(x + (WINDOW_DIM / 2) >= PIC_WIDTH)){
	//		if(!(y - (WINDOW_DIM / 2) < 0) && !(y + (WINDOW_DIM / 2) >= PIC_HEIGHT)){
			double N = WINDOW_DIM * WINDOW_DIM;

			// calc L dot 1
			double Ld1;
			windowSum(l, x, row, Ld1);
			Ld1 = Ld1 / N;

			// calc R dot 1
			double Rd1;
			windowSum(r, x + y, row, Rd1);
			Rd1 = Rd1 / N;

			// calc (L dot R) / N
			double LdR;
			winDotProduct(l, r, x, row, y, LdR);
			LdR = LdR / N;

			// calc (L dot L) / N
			double LdL;
			winDotProduct(l, l, x, row, 0, LdL);
			LdL = LdL / N;

			// calc (R dot R) / N
			double RdR;
			winDotProduct(r, r, x+y, row, 0, RdR);
			RdR = RdR / N;
			//winDotProduct(r, r, 40+y, 40, 0, LdR);

			// calculate correlation coefficient
			// [n(X.Y) - (X.1)(Y.1)] / [(n(X.X) - X.1)(n(Y.Y - Y.1))]
			double top = (WINDOW_DIM * WINDOW_DIM) * LdR - Ld1 * Rd1;
			double bot = ((WINDOW_DIM * WINDOW_DIM) * LdL - Ld1) * ((WINDOW_DIM * WINDOW_DIM) * (RdR - Rd1));

			double	corCoef = top / bot;
			__syncthreads();
			out[x + (y*PIC_WIDTH)] = y;

}

int main()
{
	//    my reference code
	//    int* helloMtx = (int*) malloc(sizeof(int) * 9);
	//    int* d_helloMtx;
	//    size_t size = 9 * sizeof(int);
	//
	//    cudaMalloc(&d_helloMtx, 9 * sizeof(int));
	//    helloWorld<<<1,9>>>(d_helloMtx);
	//    cudaMemcpy(helloMtx, d_helloMtx, size, cudaMemcpyDeviceToHost);
	//    cudaFree(&d_helloMtx);
	//    for(int i = 0; i < 9; i++){
	//        printf("%d", helloMtx[i]);
	//    }

	int* leftmtx = (int*) malloc(sizeof(int) * PIC_WIDTH * PIC_HEIGHT);
	int* rightmtx = (int*) malloc(sizeof(int) * PIC_WIDTH * PIC_HEIGHT);
	double* h_CC = (double*) malloc(sizeof(double) * PIC_WIDTH * PIC_HEIGHT); 
	for(int i = 0; i < PIC_WIDTH * PIC_HEIGHT; i++){
		leftmtx[i] = i;
		rightmtx[i] = i;
		h_CC[i] = 0.0;
	}

	int dims_2d_mtx = PIC_WIDTH * PIC_HEIGHT;
	size_t size_double_mtx = sizeof(double) *dims_2d_mtx;

	int * d_leftmtx;
	hipMalloc(&d_leftmtx, sizeof(int) * PIC_WIDTH * PIC_HEIGHT);
	int * d_rightmtx;
	hipMalloc(&d_rightmtx, sizeof(int) * PIC_WIDTH * PIC_HEIGHT);
	double * d_CC;
	hipMalloc(&d_CC, sizeof(double) * PIC_WIDTH * PIC_HEIGHT);
	// setup the 2d matrices that will hold the result of our matrix
	// mult operations, for each combination of pixels on each pixel

	hipMemcpy(d_leftmtx, leftmtx, sizeof(int) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyHostToDevice);
	hipMemcpy(d_rightmtx, rightmtx, sizeof(int) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyHostToDevice);
	hipMemcpy(d_CC, h_CC, sizeof(double) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyHostToDevice);

	dim3 threadCount(81,81);
	dim3 blockCount(1,1);
	correlationCoefficient<<<blockCount, threadCount>>>(d_leftmtx, d_rightmtx, 20, d_CC);

	hipMemcpy(h_CC, d_CC, sizeof(double) * PIC_WIDTH * PIC_HEIGHT, hipMemcpyDeviceToHost);


	for(int i = 0; i < PIC_WIDTH; i++){
		for(int j = 0; j < PIC_HEIGHT; j++){
			printf("%f ", h_CC[i + (j*PIC_WIDTH)]);
		}
		printf("\n");
	}

	hipFree(&d_leftmtx);
	hipFree(&d_rightmtx);
	hipFree(&d_CC);
	return 0;
}
